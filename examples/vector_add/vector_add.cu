/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    for (int ii = 0; ii < i % 1000; ii++) {
      if (i + ii >= numElements)
        return;
      C[i + ii] = A[i + ii] + B[i + ii];
    }
  }
}

/**
 * Host main routine
 */
extern "C" void VectorAdd(void) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

/* cudaSetDevice(0); */

  // Print the vector length to be used, and compute its size
  int numElements = 1024 * 1024;
  size_t size = numElements * sizeof(float);
  printf("[Vector addition of %d elements]\n", numElements);

  // Allocate the host input vector A
  /* float *h_A = (float *)calloc( numElements, sizeof(float)); */
  float *h_A;
  err = hipHostAlloc(&h_A, numElements * sizeof(float),
                      hipHostMallocWriteCombined);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Allocate the host input vector B
  float *h_B = (float *)calloc(numElements, sizeof(float));

  // Allocate the host output vector C
  float *h_C = (float *)calloc(numElements, sizeof(float));

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    return;
  }

  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 1024;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
  //        threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  err = hipGetLastError();
  hipDeviceSynchronize();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    return;
  }

  // Free host memory
  hipFree(h_A);
  free(h_B);
  free(h_C);

  printf("Done\n");
  return;
}
